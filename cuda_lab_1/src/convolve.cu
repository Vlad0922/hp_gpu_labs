#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <sstream>
#include <tuple>

#include <assert.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "matrix.h"

const size_t BLOCK_X = 16;
const size_t BLOCK_Y = 16;

__global__
void convolve(const float *A, const float *B, float *dst, const int matrix_size, const int kernel_size)
{
    __shared__ float s[BLOCK_X*BLOCK_Y];

    int i = (blockIdx.x*blockDim.x) + threadIdx.x;
    int j = (blockIdx.y*blockDim.y) + threadIdx.y;

    if(i < matrix_size && j < matrix_size)
    {
        float res = 0.;
        const int HM = (kernel_size - 1)/2;

        for(int k = -HM; k <= HM; ++k)
        {
            for(int l = -HM; l <= HM; ++l)
            {
                int x = i + k;
                int y = j + l;

                int ker_x = k + HM;
                int ker_y = l + HM;

                if( x >= 0 && x < matrix_size && y >= 0 && y < matrix_size)
                {
                    res += A[y*matrix_size + x]*B[ker_y*kernel_size + ker_x];
                }
            }
        }

        s[threadIdx.y*BLOCK_X + threadIdx.x] = res;
    }

    __syncthreads();

    if(i < matrix_size && j < matrix_size)
    {
        dst[j*matrix_size+i] = s[threadIdx.y*BLOCK_X + threadIdx.x];
    }
}


SquareMatrix convolve_with_cuda(const SquareMatrix &A, const SquareMatrix &B)
{
    const size_t GRID_X = A.size()/BLOCK_X + int( (A.size() % BLOCK_X) != 0);
    const size_t GRID_Y = A.size()/BLOCK_Y + int( (A.size() % BLOCK_Y) != 0);

    const dim3 blockSize(BLOCK_X, BLOCK_Y, 1);  
    const dim3 gridSize(GRID_X, GRID_Y, 1); 

    SquareMatrix C(A.size());

    float *dev_A;
    float *dev_kernel;
    float *dev_result;

    hipMalloc((void **)&dev_result, A.size()*A.size()*sizeof(float));
    hipMalloc((void **)&dev_A, A.size()*A.size()*sizeof(float));
    hipMalloc((void **)&dev_kernel, B.size()*B.size()*sizeof(float));  

    hipMemcpy(dev_A, A.data(), A.size()*A.size()*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_kernel, B.data(), B.size()*B.size()*sizeof(float), hipMemcpyHostToDevice);

    convolve<<<gridSize, blockSize>>>(dev_A, dev_kernel, dev_result, A.size(), B.size());

    hipMemcpy(C.data(), dev_result, C.size()*C.size()*sizeof(float), hipMemcpyDeviceToHost);

    hipFree(dev_A);
    hipFree(dev_kernel);
    hipFree(dev_result);

    return C;
}

const char *INPUT_FNAME  = "input.txt";
const char *OUTPUT_FNAME = "output.txt";

void read_from_fstream(std::ifstream &input, float *arr, const size_t size)
{
    float val;
    for(size_t i = 0; i < size; ++i)
    {
        for(size_t j = 0; j < size; ++j)
        {
            input >> val;
            arr[i*size + j] = val;
        }
    }
}

std::tuple<SquareMatrix, SquareMatrix> read_data(const char *fname)
{
    std::ifstream input_file(fname);

    if(!input_file.is_open())
    {
        std::cerr << "Cannot open " << fname << "!\n";
        exit(1);
    }

    size_t N, M;

    input_file >> N >> M;

    SquareMatrix A(N);
    SquareMatrix B(M);

    read_from_fstream(input_file, A.data(), N);
    read_from_fstream(input_file, B.data(), M);

    return std::make_tuple(A, B);
}

void print_matrix(const SquareMatrix &m)
{
    for(size_t i = 0; i < m.size(); ++i)
    {
        for(size_t j = 0; j < m.size(); ++j)
        {
            std::cout << m[i][j] << ' ';
        }
        std::cout << '\n';
    }
}


float calc_cell_convolve(const SquareMatrix &A, const SquareMatrix &B, int i, int j)
{
    float val = 0;
    int HM = (B.size() - 1)/2;

    for(int k = -HM; k <= HM; ++k)
    {
        for(int l = -HM; l <= HM; ++l)
        {
            val += A.get_val(i+k, j+l)*B.get_val(k+HM, l+HM);
        }
    }

    return val;
}


// simple function to check the result
SquareMatrix convolve(const SquareMatrix &A, const SquareMatrix &B)
{
    SquareMatrix C(A.size());

    for(int i = 0; i < A.size(); ++i)
    {
        for(int j = 0; j < A.size(); ++j)
        {
            C[i][j] = calc_cell_convolve(A, B, i, j);
        }
    }

    return C;
}


void write_data(const char *fname, const SquareMatrix &m)
{
    std::ofstream out_file(fname);

    if(!out_file.is_open())
    {        
        std::cerr << "Cannot open " << fname << "!\n";
        return;
    }    

    for(size_t i = 0; i < m.size(); ++i)
    {
        for(size_t j = 0; j < m.size(); ++j)
        {
            out_file << m[j][i] << ' ';
        }
        out_file << '\n';
    }
}


void run_test()
{
    {
        SquareMatrix A(1024);
        SquareMatrix B(3);

        A.fill(1.);
        B.fill(1.);

        SquareMatrix correct = convolve(A, B);
        SquareMatrix cuda_res = convolve_with_cuda(A, B);

        assert(correct == cuda_res && "1st test didn't pass");
    }

    {
        SquareMatrix A(1024);
        SquareMatrix B(9);

        A.fill(1.);
        B.fill(1.);

        SquareMatrix correct = convolve(A, B);
        SquareMatrix cuda_res = convolve_with_cuda(A, B);

        assert(correct == cuda_res && "2nd test didn't pass");
    }

    {
        SquareMatrix A(1);
        SquareMatrix B(9);

        A.fill(1.);
        B.fill(1.);

        SquareMatrix correct = convolve(A, B);
        SquareMatrix cuda_res = convolve_with_cuda(A, B);

        assert(correct == cuda_res && "3rd test didn't pass");
    }

    {
        SquareMatrix A(31);
        SquareMatrix B(9);

        A.fill(1.);
        B.fill(1.);

        SquareMatrix correct = convolve(A, B);
        SquareMatrix cuda_res = convolve_with_cuda(A, B);

        assert(correct == cuda_res && "4th test didn't pass");
    }

    {
        SquareMatrix A(1023);
        SquareMatrix B(9);

        A.fill(1.);
        B.fill(1.);

        SquareMatrix correct = convolve(A, B);
        SquareMatrix cuda_res = convolve_with_cuda(A, B);

        assert(correct == cuda_res && "5th test didn't pass");
    }

    std::cout << "All tests passed!\n";
}


int main(int argc, char **argv)
{
    if(argc == 2 && strcmp(argv[1], "test") == 0)
    {
        run_test();
    }
    else
    {
        SquareMatrix A, B;
        std::tie(A, B) = read_data(INPUT_FNAME);

        SquareMatrix correct = convolve(A, B);
        SquareMatrix cuda_res = convolve_with_cuda(A, B);
        
        if(correct == cuda_res)
        {
            std::cout << "Ok!\n";
            
        }
        else
        {
            std::cout << "Error!\n";
        }

        write_data(OUTPUT_FNAME, cuda_res);
    }


    return 0;
}
