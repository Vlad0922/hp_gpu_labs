#include "hip/hip_runtime.h"


__global__ void convolve(const float *A, const float *B, float *dst, const int kernel_size)
{

}


SquareMatrix convolve_with_cuda(const SquareMatrix &A, const SquareMatrix &B)
{
	SquareMatrix C(A.size());

	float *dev_A;
	float *dev_kernel;
	float *dev_result;
	int kernel_size;	

	hipMalloc((void **)&dev_result, A.size()*A.size()*sizeof(float));
	hipMalloc((void **)&dev_A, A.size()*A.size()*sizeof(float));
	hipMalloc((void **)&dev_kernel, B.size()*B.size()*sizeof(float));	

	hipMemcpy(dev_A, A.data(), A.size()*A.size()*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_kernel, B.data(), B.size()*B.size()*sizeof(float), hipMemcpyHostToDevice);

	convolve<<<8,8>>>(dev_A, dev_kernel, dev_result)

	hipMemcpy(C.data(), dev_result, C.size()*C.size()*sizeof(float), hipMemcpyDeviceToHost);

	hipFree(dev_A);
	hipFree(dev_kernel);
	hipFree(dev_result);


	return C;
}